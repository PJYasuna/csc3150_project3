#include "hip/hip_runtime.h"
﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000; 
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;
    vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = 0;
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;                  /* page size = 32b */
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;    /* 16KB for page table setting */
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;  /* 32KB for physical memory */
  vm->STORAGE_SIZE = STORAGE_SIZE;          /* 128KB for the disk storage */
  vm->PAGE_ENTRIES = PAGE_ENTRIES;          /* = PHYSICAL_MEM_SIZE / PAGE_SIZE =  1024*/

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ int find_frame_number(VirtualMemory *vm, int target) {
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    if (target == vm->invert_page_table[i + vm->PAGE_ENTRIES]&& vm->invert_page_table[i] != 0x80000000) {
      for (int j = 0; j < vm->PAGE_ENTRIES; j++){
        vm->invert_page_table[j + 2 * vm->PAGE_ENTRIES] += 1;
      }
      return i;
    }
  }
  return -1;
}

__device__ int get_LRU_position(VirtualMemory *vm) {
  // empty
  int max = vm->invert_page_table[0 + 2 * vm->PAGE_ENTRIES];
  int res = 0;
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    if (vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] > max)  {
      max = vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES];
      res = i;
    }
  }
  return res;
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {

  __syncthreads();
	if (addr % 4 != ((int)threadIdx.x)) return;
  /* Complete vm_write function to write value into data buffer */
  int pid_vpn = addr / vm->PAGESIZE;      
  int offset = addr % vm->PAGESIZE;          
  int frame_number = find_frame_number(vm, pid_vpn);
  uchar res;

  if (frame_number != -1){ // exist
    vm->invert_page_table[frame_number + 2 * vm->PAGE_ENTRIES] = 0;
    res = vm->buffer[frame_number*vm->PAGESIZE+offset];           
  }

  else{ // not exist
    *vm->pagefault_num_ptr += 1;

    int empty = -1; // to find if there exist empty idx
    for (int i = 0; i < vm->PAGE_ENTRIES; i++){  
      if (vm->invert_page_table[i] == 0x80000000){ 
          empty = i;               
          break;
      }
    }

    if (empty != -1) { // empty
      for (int i = 0; i < vm->PAGESIZE; i++){
        vm->buffer[empty*vm->PAGESIZE+i] = vm->storage[pid_vpn*vm->PAGESIZE+i]; // swap in
      }
      // update page table
      vm->invert_page_table[empty+vm->PAGE_ENTRIES] = pid_vpn;
      vm->invert_page_table[empty] = 0; 
      vm->invert_page_table[empty + 2 * vm->PAGE_ENTRIES] = 0; // update lru table
      res = vm->buffer[empty*vm->PAGESIZE+offset];
    }

    else { // no empty, so find lru
      int LRU_idx = get_LRU_position(vm); // victim
      int LRU_disk_idx = vm->invert_page_table[LRU_idx + vm->PAGE_ENTRIES];
      for (int i = 0; i < vm->PAGESIZE; i++){                                            
        vm->storage[LRU_disk_idx*vm->PAGESIZE+i] = vm->buffer[LRU_idx*vm->PAGESIZE+i];  // swap out
        vm->buffer[LRU_idx*vm->PAGESIZE+i] = vm->storage[pid_vpn*vm->PAGESIZE+i]; // swap in
      }
      // update page table
      vm->invert_page_table[LRU_idx+vm->PAGE_ENTRIES] = pid_vpn;
      vm->invert_page_table[LRU_idx] = 0; 
      vm->invert_page_table[LRU_idx + 2 * vm->PAGE_ENTRIES] = 0; // update lru table
      res = vm->buffer[LRU_idx*vm->PAGESIZE+offset];
    }

  }

  return res;
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  __syncthreads();
	if (addr % 4 != ((int)threadIdx.x)) return;
  /* Complete vm_write function to write value into data buffer */
  int pid_vpn = addr / vm->PAGESIZE;      
  int offset = addr % vm->PAGESIZE;           /* get the last 5 bits */
  int frame_number = find_frame_number(vm, pid_vpn);

  if (frame_number != -1) {
    vm->invert_page_table[frame_number + 2 * vm->PAGE_ENTRIES] = 0;
    vm->buffer[frame_number*vm->PAGESIZE+offset] = value;
  }

  else{ 
    *vm->pagefault_num_ptr += 1;

    int empty = -1;
    for (int i = 0; i < vm->PAGE_ENTRIES; i++){  
      if (vm->invert_page_table[i] == 0x80000000){ 
          empty = i;               
          break;
      }
    }

    if (empty != -1) {
      for (int i = 0; i < vm->PAGESIZE; i++){
        vm->buffer[empty*vm->PAGESIZE+i] = vm->storage[pid_vpn*vm->PAGESIZE+i]; 
      }

      vm->invert_page_table[empty+vm->PAGE_ENTRIES] = pid_vpn;
      vm->invert_page_table[empty] = 0; 
      vm->invert_page_table[empty + 2 * vm->PAGE_ENTRIES] = 0;
      vm->buffer[empty*vm->PAGESIZE+offset] = value;
    }

    else {
      int LRU_idx = get_LRU_position(vm); // victim
      int LRU_disk_idx = vm->invert_page_table[LRU_idx + vm->PAGE_ENTRIES]; // pid_vpn
      for (int i = 0; i < vm->PAGESIZE; i++){                                              
        vm->storage[LRU_disk_idx*vm->PAGESIZE+i] = vm->buffer[LRU_idx*vm->PAGESIZE+i];   
        vm->buffer[LRU_idx*vm->PAGESIZE+i] = vm->storage[pid_vpn*vm->PAGESIZE+i]; 
      }
      vm->invert_page_table[LRU_idx+vm->PAGE_ENTRIES] = pid_vpn;
      vm->invert_page_table[LRU_idx] = 0; 
      vm->invert_page_table[LRU_idx + 2 * vm->PAGE_ENTRIES] = 0; 
      vm->buffer[LRU_idx*vm->PAGESIZE+offset] = value;
    }
  }
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
  for (int i = 0; i < input_size / 4; i++){
    results[i * 4 + (int)threadIdx.x] = vm_read(vm, i * 4 + (int)threadIdx.x+offset);
  }
}